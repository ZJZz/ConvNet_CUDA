#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <string.h>
#include <float.h>
#include <math.h>
#include <assert.h>
#include <xmmintrin.h>
#include <immintrin.h>

#include "hipDNN.h"
#include "util.h"
#include "Kernel128_winograd.h"


#define cudaCheckError() {																\
	hipError_t e=hipGetLastError();													\
	if(e!=hipSuccess) {																\
		printf("Cuda failure %s:%d:'%s'\n",__FILE__,__LINE__,hipGetErrorString(e));	\
		exit(EXIT_FAILURE);																\
	}																					\
}`

#define MY_KERNEL 1

// 2^7 = 128 | 6 * 128 = 768
#define d(input, i, j, Inz) ( input[Inz + i*768 + (j<<7)] )

__global__ void kernel_128_winograd_BtdB(float *pInputs, float *pOutputs) {

	// grid size     = (4, 4)
	// block size    = (row: 6 col: 128)
	// shared memory size = (6 * 6 * 128)
	
	// pInputs  (16 * 16 * 128)
	// pOutputs (16 *  6 *   6 * 128)

    int Inx  = blockIdx.x<<2; // 0 4 8 12
    int Iny0 = blockIdx.y<<2; // 0 4 8 12
    int Iny1 = threadIdx.y;   // 0~5   row in block, local
    int Inz  = threadIdx.x;   // 0~127 col in block 
    int Iny  = Iny0+Iny1;     // 0~5 | 4~9 | 8~13 | 12~17
    int stride_r = 2048;      // 16 * 128
    int stride_c = 128;       // 2048 = 16*128 may relate with channel number
    int c_glb_start = Inx*stride_r + Iny*stride_c + Inz;
    int c_input = Iny1 * stride_c + Inz;

	extern __shared__ float input[];

	int tmp[6] = {0, 768, 1536, 2304, 3072, 3840}; // 768 = 6*128
	for (int i = 0; i < 6; i++) {
		// copy value to shared memory
		input[c_input + tmp[i]] = pInputs[c_glb_start + i*stride_r]; // 共享内存的起点
	}
	__syncthreads();

	float BTd[6]; // one row of BTd, the row number bind with Iny1
	switch(Iny1) {
		case 0:
			for (int j = 0; j < 6; j++) {
				BTd[j] = d(input, 0, j, Inz)*4 - d(input, 2, j, Inz)*5 + d(input, 4, j, Inz);
			}
			break;
		case 1:
			for (int j = 0; j < 6; j++) {
				BTd[j] = -d(input, 1, j, Inz)*4 - d(input, 2, j, Inz)*4 + d(input, 3, j, Inz) + d(input, 4, j, Inz);
			}
			break;
		case 2:
			for (int j = 0; j < 6; j++) {
				BTd[j] = d(input, 1, j, Inz)*4 - d(input, 2, j, Inz)*4 - d(input, 3, j, Inz) + d(input, 4, j, Inz);
			}
			break;
		case 3:
			for (int j = 0; j < 6; j++) {
				BTd[j] = -d(input, 1, j, Inz)*2 - d(input, 2, j, Inz) + d(input, 3, j, Inz)*2 + d(input, 4, j, Inz);
			}
			break;
		case 4:
			for (int j = 0; j < 6; j++) {
				BTd[j] = d(input, 1, j, Inz)*2 - d(input, 2, j, Inz) - d(input, 3, j, Inz)*2 + d(input, 4, j, Inz);
			}
			break;
		case 5:
			for (int j = 0; j < 6; j++) {
				BTd[j] = d(input, 1, j, Inz)*4 - d(input, 3, j, Inz)*5 + d(input, 5, j, Inz);
			}
			break;
	}
	__syncthreads();

	// Iny1  0~5   row in block
    // Inz   0~127 col in block 
	int tmp_offset = Iny1*768+Inz; // 768 = 128 * 6
	for (int i = 0; i < 6; i++) {
		// stride_c = 128
		input[tmp_offset + i*stride_c] = BTd[i];
	}
	__syncthreads();

	float BTdB[6]; // one row of BTdB, the row number bind with Iny1
	switch(Iny1) {
		case 0:
			for (int i = 0; i < 6; i++) { // iterate col in d
				BTdB[i] = 4*d(input, i, 0, Inz) - 5*d(input, i, 2, Inz) + d(input, i, 4, Inz);
			}
			break;
		case 1:
			for (int i = 0; i < 6; i++) {
				BTdB[i] = -4*d(input, i, 1, Inz) - 4*d(input, i, 2, Inz) + d(input, i, 3, Inz) + d(input, i, 4, Inz);
			}
			break;
		case 2:
			for (int i = 0; i < 6; i++) {
				BTdB[i] = 4*d(input, i, 1, Inz) - 4*d(input, i, 2, Inz) - d(input, i, 3, Inz) + d(input, i, 4, Inz);
			}
			break;
		case 3:
			for (int i = 0; i < 6; i++) {
				BTdB[i] = -2*d(input, i, 1, Inz) - d(input, i, 2, Inz) + 2*d(input, i, 3, Inz) + d(input, i, 4, Inz);
			}
			break;
		case 4:
			for (int i = 0; i < 6; i++) {
				BTdB[i] = 2*d(input, i, 1, Inz) - d(input, i, 2, Inz) - 2*d(input, i, 3, Inz) + d(input, i, 4, Inz);
			}
			break;
		case 5:
			for (int i = 0; i < 6; i++) {
				BTdB[i] = 4*d(input, i, 1, Inz) - 5*d(input, i, 3, Inz) + d(input, i, 5, Inz);
			}
			break;
	}
	__syncthreads();

	for (int i = 0; i < 6; i++) {
		// Iny1: 0~5   row in block
		// Inz : 0~127 col in block
		// 2048 = 16 * 128
		// pOutputs (16 * 6 * 6 * 128)
		pOutputs[(Iny1 + i*6)*2048 + (blockIdx.x*4+blockIdx.y)*128 + Inz] = BTdB[i]; // check
		// (blockIdx.x*4+blockIdx.y)*128 responsible for 16 tile
	}
}


__global__ void kernel_128_winograd_AtIA(float *pInputs, float *pBiases, float *pScales, float *pOutputs) {
	
	int Tilex = blockIdx.x;
	int Tiley = blockIdx.y;
	int kz    = blockIdx.z;
	int Iny   = threadIdx.y;
	int Inx   = threadIdx.x;
	
	int c_input = Inx*6 + Iny;

	__shared__ float bias, scale;
	extern __shared__ float input[];

	// copy to shared memory
	input[c_input] = pInputs[c_input*16*128 + (Tilex*4+Tiley)*128 + kz];
	bias = pBiases[kz];
	scale = pScales[kz];
	__syncthreads();

	float tmp = 0;
	switch(Inx) {
		case 0:
			tmp = input[Iny] + input[6+Iny] + input[12+Iny] + input[18+Iny] + input[24+Iny];
			break;
		case 1:
			tmp = input[6+Iny] - input[12+Iny] + 2*input[18+Iny] - 2*input[24+Iny];
			break;
		case 2:
			tmp = input[6+Iny] + input[12+Iny] + 4*input[18+Iny] + 4*input[24+Iny];
			break;
		case 3:
			tmp = input[6+Iny] - input[12+Iny] + 8*input[18+Iny] - 8*input[24+Iny] + input[30+Iny];
			break;
	}
	__syncthreads();

	input[c_input] = tmp;
	__syncthreads(); // finish AT * C

	
	if (Inx > 3 || (Tilex == 3 && Inx > 1)) return;


	// read Because when Inx > 3 the data are useless
	
	// may relate with write: Tilex = 3, Inx = 0
	
	int x;
	float o;
	switch(Iny) {
		// 128 = 8 *16
		// bug ? ( ( (TileX<<2) + InX ) * 16 + (TileY<<2) + 0 ) * 128 + kz
		case 0:
			x = Inx*6;
			o = scale*(input[x]+input[x+1]+input[x+2]+input[x+3]+input[x+4])+ bias;
			pOutputs[ ( ( (Tilex<<2) + 1 + Inx ) * 16 + (Tiley<<2) + 1 ) * 128 + kz] = o > 0 ? o : 0;
			break;
		case 1:
			x = Inx*6;
			o = scale*(input[x+1] - input[x+2] + 2*input[x+3] - 2*input[x+4]) + bias;
			pOutputs[ ( ( (Tilex<<2) + 1 + Inx ) * 16 + (Tiley<<2) + 2 ) * 128 + kz] = o > 0 ? o : 0;
			break;
		case 2:
			if (Tiley == 3) break;
			x = Inx*6;
			o = scale*(input[x+1] + input[x+2] + 4*input[x+3] + 4*input[x+4]) + bias;
			pOutputs[ ( ( (Tilex<<2) + 1 + Inx ) * 16 + (Tiley<<2) + 3 ) * 128 + kz] = o > 0 ? o : 0;
			break;
		case 3:
			if (Tiley == 3) break;
			x = Inx*6;
			o = scale*(input[x+1] - input[x+2] + 8*input[x+3] - 8*input[x+4] + input[x+5]) + bias;
			pOutputs[ ( ( (Tilex<<2) + 1 + Inx ) * 16 + (Tiley<<2) + 4 ) * 128 + kz] = o > 0 ? o : 0;
			break;
	}
}


__global__ void kernel_128_OuterProduct_128(float *A, float *B, float *C) {
    
    int Tile = blockIdx.x; // one of 36 matrix 
    int Part = blockIdx.y; // one of 128 * (8 + 8)
    int tX   = threadIdx.x; // col number inside a submatrix (8 * 128)
    int tY   = threadIdx.y; // row number inside a submatrix (8 * 128)

    
    // 2^10 = 1024 | 2^11 = 2048 | 2^14 = 2^7 * 2^7 = 16384 | 2^10 = 1024
    int c_input  = tY * 128 + tX; // compute ordinal number in a (8 * 128) 
	int T_offset = (Tile<<11) + (Part<<10) + c_input; // A's location
	int c_kernel = c_input; // input offset in kernel 
    int B_offset = (Tile<<14) + c_kernel; // 2^7 * 2^7 = 128 * 128
	
	extern __shared__ float input[];
    float *kernel = input + 1024; // after 1024 one block thread // check
    float *out    = kernel + 8192;  // 64 * 128  = 8192 (one half of B) // check
	int B_stride[32] = {0, 128, 256, 384, 512, 640, 768, 896, 1024, 1152, 1280, 1408, 1536, 1664, 1792, 1920, 2048, 2176, 2304, 2432, 2560, 2688, 2816, 2944, 3072, 3200, 3328, 3456, 3584, 3712, 3840, 3968};//, 4096, 4224, 4352, 4480, 4608, 4736, 4864, 4992, 5120, 5248, 5376, 5504, 5632, 5760, 5888, 6016, 6144, 6272, 6400, 6528, 6656, 6784, 6912, 7040, 7168, 7296, 7424, 7552, 7680, 7808, 7936, 8064};
	out[c_input] = 0.0f;  // check

	// copy global memory of A (8 * 128) to shared memory
    input[c_input] = A[T_offset]; // check 
    

    // outer product
	for (int k = 0; k < 4; k++) { // blocking 
		int B_start = B_offset + (k<<12); // 32*64 | 2^12 = 4096 
		
		// copy  global memory of B (64 * 128) to shared memory
		// kernel (128 * 64) B (128 *128) 
		kernel[c_kernel]      = B[B_start]; // 128 * 0
        kernel[c_kernel+1024] = B[B_start+1024]; // 128 * 8 = 64 * 16
        kernel[c_kernel+2048] = B[B_start+2048]; // 128 * 16
		kernel[c_kernel+3072] = B[B_start+3072]; // 128 * 24
		// copy 4 times, total 32 channel long

		__syncthreads();

		float sum = 0;
		int y_tmp = (tY<<7)+(k<<5); // 2^7 = 128 | 2^5 = 32 // horzital direction
		for (int j = 0; j < 32; j++) {
			sum += input[y_tmp + j] * kernel[tX + B_stride[j]]; // B_stride[j] vertical postion
		}
		out[tY*128 + tX] += sum;
		__syncthreads();
	}

	// copy shared memory of C (8 * 128) to global memory
	C[T_offset] = out[c_input];
}

int kernel_128() {
	float *input_ = get_parameter(inputName128, 16*16*128);
	float *bias = get_parameter(biasName128, 128);
	float *input, *output, *l_weights, *l_bias;
	uint64_t nT1 = 0, nT2 = 0, nT1_cudnn = 0, nT2_cudnn = 0;
	hipError_t s;

	/////////////////////////////////

	// My Kernel

	/////////////////////////////////


	/*  1. Data preparation  */
	float *t_input, *ip;
	//float *kernel = get_Winograd_Kernel128(weight_winograd_Name128, 128);
	float *kernel = get_parameter(weight_winograd_Name128, 36*128*128);
	float *l_bnBias, *l_bnScale, *bnBias, *bnScale;

	int nInput = 16*16*128, nOutput = 16*16*128, nWeights = 36*128*128, nBias = 128, nTransInput = 16*6*6*128, nInnerProd = 16*6*6*128;
	hipMalloc((void **) &input, nInput<<3);
	hipMalloc((void **) &output, nOutput<<2);
	hipMalloc((void **) &l_weights, nWeights<<2);
	hipMalloc((void **) &l_bias, nBias<<2);
	hipMalloc((void **) &t_input, nTransInput<<2);
	hipMalloc((void **) &ip, nInnerProd<<2);
    
    hipMemset((void *) input, 0, nInput<<3);
	hipMemset((void *) output, 0, nOutput<<2);
	hipMemset((void *) t_input, 0, nTransInput<<2);
	hipMemset((void *) l_weights, 0, nWeights<<2);
	hipMemset((void *) ip, 0, nInnerProd<<2);
    
    hipMemcpy(input, input_, nInput<<2, hipMemcpyHostToDevice);
	hipMemcpy(l_weights, kernel, nWeights<<2, hipMemcpyHostToDevice);
	hipMemcpy(l_bias, bias, nBias<<2, hipMemcpyHostToDevice);
	
	bnBias = get_parameter(bnBias_winograd_Name128, 128);
	bnScale = get_parameter(bnScale_winograd_Name128, 128);
    
    hipMalloc((void **) &l_bnBias, nBias<<2);
	hipMalloc((void **) &l_bnScale, nBias<<2);
    
    hipMemcpy(l_bnBias, bnBias, nBias<<2, hipMemcpyHostToDevice);
	hipMemcpy(l_bnScale, bnScale, nBias<<2, hipMemcpyHostToDevice);
    
    float tmp_winograd[nOutput];

	
	/*  2. Computing  */
	nT1 = getTimeMicroseconds64();

	kernel_128_winograd_BtdB <<<dim3(4, 4), dim3(128, 6), (6*6*128)<<2 >>> (input, t_input);
	kernel_128_OuterProduct_128<<<dim3(36, 2), dim3(128, 8), (8*128 + 64*128 + 8*128)<<2 >>> (t_input, l_weights, ip);
	kernel_128_winograd_AtIA <<<dim3(4, 4, 128), dim3(6, 6), ((6*6)<<2)>>> (ip, l_bnBias, l_bnScale, output);
	//cudaCheckError();
	hipDeviceSynchronize();
	
	nT2 = getTimeMicroseconds64();
	printf("TotalTime = %d us\n", nT2-nT1); 


	/*  3. Copy back and free  */
	s = hipMemcpy(tmp_winograd, output, nOutput<<2, hipMemcpyDeviceToHost);
	printf("%s\n", hipGetErrorName(s));
	//cudaCheckError();

	hipFree(t_input);
	hipFree(output);
	hipFree(l_weights);
	hipFree(l_bias);
	hipFree(ip);

	free(kernel);
	free(bnScale);
	free(bnBias);


	/////////////////////////////////

	// cuDNN

	/////////////////////////////////

	/*  1. Data preparation  */
	kernel = get_parameter(weight_NCHW_Name128, 9*128*128);
	bnBias = get_parameter(bnBiasName128, 128);
	bnScale = get_parameter(bnScaleName128, 128);
	float* eMean = get_parameter(eMeanName128, 128);
	float* eVar = get_parameter(eVarName128, 128);
	float *l_eMean, *l_eVar;
	nInput = 16*16*128, nOutput = 14*14*128, nWeights = 3*3*128*128, nBias = 128;

	hipMalloc((void **) &output, nOutput<<2);
	hipMalloc((void **) &l_weights, nWeights<<2);
	hipMalloc((void **) &l_bias, nBias<<2);
    
    hipMemcpy(l_weights, kernel, nWeights<<2, hipMemcpyHostToDevice);
	hipMemcpy(l_bias, bias, nBias<<2, hipMemcpyHostToDevice);

	hipMalloc((void **) &l_eMean, nBias<<2);
	hipMalloc((void **) &l_eVar, nBias<<2);
    
    hipMemcpy(l_bnBias, bnBias, nBias<<2, hipMemcpyHostToDevice);
	hipMemcpy(l_bnScale, bnScale, nBias<<2, hipMemcpyHostToDevice);
	hipMemcpy(l_eMean, eMean, nBias<<2, hipMemcpyHostToDevice);
	hipMemcpy(l_eVar, eVar, nBias<<2, hipMemcpyHostToDevice);

	hipMemset((void *) output, 0, nOutput<<2);

	float tmp_cudnn[nOutput];


	/*  2. cuDNN preparation  */
	hipdnnStatus_t status;
	float one = 1.0, zero = 0.0;
	int size;

	hipdnnHandle_t handle;
	status = hipdnnCreate(&handle);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed1\n");

	hipdnnTensorDescriptor_t xdesc, ydesc, bdesc;
	hipdnnFilterDescriptor_t wdesc; // HIPDNN_TENSOR_NHWC, HIPDNN_TENSOR_NCHW
	status = hipdnnCreateTensorDescriptor(&xdesc);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed2\n");
	status = hipdnnSetTensor4dDescriptor(xdesc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_FLOAT, 1, 128, 16, 16);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed3\n");
	status = hipdnnCreateTensorDescriptor(&ydesc);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed4\n");
	status = hipdnnSetTensor4dDescriptor(ydesc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_FLOAT, 1, 128, 14, 14);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed5\n");
	status = hipdnnCreateFilterDescriptor(&wdesc);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed6\n");
	status = hipdnnSetFilter4dDescriptor(wdesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, 128, 128, 3, 3);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed7\n");
	status = hipdnnCreateTensorDescriptor(&bdesc);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed8\n");
	status = hipdnnSetTensor4dDescriptor(bdesc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_FLOAT, 1, 128, 1, 1);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed9\n");
	hipdnnConvolutionDescriptor_t conv_desc;
	status = hipdnnCreateConvolutionDescriptor(&conv_desc);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed10\n");
	status = hipdnnSetConvolution2dDescriptor(conv_desc, 0,0, 1,1,1,1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT); //HIPDNN_CONVOLUTION
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed11\n");

    // Activation
	hipdnnActivationDescriptor_t act_desc;
	status = hipdnnCreateActivationDescriptor(&act_desc);  
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed12\n");
	status = hipdnnSetActivationDescriptor(act_desc, HIPDNN_ACTIVATION_RELU, HIPDNN_NOT_PROPAGATE_NAN, 0);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed13\n");

    // BN
	hipdnnTensorDescriptor_t bnScaleBiasMeanVarDesc;
	status = hipdnnCreateTensorDescriptor(&bnScaleBiasMeanVarDesc);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed14\n");
	status = hipdnnSetTensor4dDescriptor(bnScaleBiasMeanVarDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 128, 1, 1);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed15\n");

	hipdnnConvolutionFwdAlgo_t algo = (hipdnnConvolutionFwdAlgo_t)6;

	status = hipdnnGetConvolutionForwardWorkspaceSize(handle,
	   xdesc,
	   wdesc,
	   conv_desc,
	   ydesc,
	   algo,
	   (size_t *)&(size));

	float *extra;
	hipMalloc((void **) &extra, size);


	/*  3. Computing  */
	nT1_cudnn = getTimeMicroseconds64();

	status = hipdnnConvolutionForward(handle, &one,
		xdesc, input, wdesc, l_weights, 
		conv_desc, algo, 
		extra, size, &zero,
		ydesc, output);
	if (status != HIPDNN_STATUS_SUCCESS) printf("Not Successed1\n");

	status = hipdnnBatchNormalizationForwardInference(handle, HIPDNN_BATCHNORM_SPATIAL,
		&one, &zero, 
		ydesc, output, ydesc, output,
		bnScaleBiasMeanVarDesc, l_bnScale, l_bnBias, l_eMean, l_eVar, HIPDNN_BN_MIN_EPSILON);
	if (status != HIPDNN_STATUS_SUCCESS) printf("Not Successed2\n");

	status = hipdnnActivationForward(handle, act_desc, &one,
		ydesc, output, &zero,
		ydesc, output);
	if (status != HIPDNN_STATUS_SUCCESS) printf("Not Successed3\n");

	hipDeviceSynchronize();
	nT2_cudnn = getTimeMicroseconds64();
	printf("cuDNN TotalTime = %d us\n", nT2_cudnn-nT1_cudnn);


	/*  4. Copy back and free  */
	s = hipMemcpy(tmp_cudnn, output, nOutput<<2, hipMemcpyDeviceToHost);
	printf("%s\n", hipGetErrorName(s));

	hipFree(extra);
	hipFree(input);
	hipFree(output);
	hipFree(l_weights);
	hipFree(l_bias);

	hipFree(l_bnScale);
	hipFree(l_bnBias);
	hipFree(l_eMean);
	hipFree(l_eVar);

	free(bias);
	free(kernel);

	free(bnScale);
	free(bnBias);
	free(eMean);
	free(eVar);
	free(input_);

	output_checker(tmp_winograd, tmp_cudnn, 14, 128, 1);

	return ((nT2-nT1) << 16) | (nT2_cudnn-nT1_cudnn);
}