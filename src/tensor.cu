#include "tensor.h"
#include <hip/hip_runtime.h>

Tensor::Tensor(int batch_num, int channel, int height, int width):
    shape_(batch_num, channel, height, width),d_data_(nullptr), h_data_(nullptr),device_allocated_(false), host_allocated_(false)
{
	allocateMemoryIfNotAllocated(shape_);
}

Tensor::Tensor(Shape shape):Tensor(shape.n_, shape.c_, shape.h_, shape.w_)
{ }


void Tensor::allocateMemoryIfNotAllocated(Shape& shape)
{
    if(!device_allocated_ && !host_allocated_)
    {
        shape_ = shape;
        allocateMemory();
    }
}

void Tensor::allocateMemory()
{
    allocateHostMemory();
    allocateCudaMemory();
}

void Tensor::allocateHostMemory()
{
    if(!host_allocated_)
    {
        h_data_ = std::shared_ptr<float>(new float[shape_.total_elements()], [&](float *ptr){  delete[] ptr; });
        host_allocated_ = true;
    }
}

void Tensor::allocateCudaMemory()
{
    if(!device_allocated_)
    {
        float *device_mem = nullptr;
        hipMalloc(&device_mem, shape_.total_elements() * sizeof(float));
        // TODO: add some exception handle
        d_data_ = std::shared_ptr<float>(device_mem, [&](float *ptr){ hipFree(ptr); });
        device_allocated_ = true;
    }
}

void Tensor::transfer_H2D()
{
    if(device_allocated_ && host_allocated_)
    {
        // get() ?
        hipMemcpy(h_data_.get(), d_data_.get(), shape_.total_elements() * sizeof(float), hipMemcpyHostToDevice);
        // TODO: add some exception handle
    }
    else
    {
        printf("Cannot copy host data to not allocated memory on device\n");
    }
}

void Tensor::transfer_D2H()
{
    if(device_allocated_ && host_allocated_)
    {
        // get() ?
        hipMemcpy(h_data_.get(), d_data_.get(), shape_.total_elements() * sizeof(float), hipMemcpyDeviceToHost);
        // TODO: add some exception handle
    }
    else
    {
        printf("Cannot copy device data to not allocated memory on host\n");
    }
}

float& Tensor::operator[](const int index)
{
	return h_data_.get()[index];
}

const float& Tensor::operator[](const int index) const
{
	return h_data_.get()[index];
}

void Tensor::print_tensor(std::string name, bool view_param = false, int num_batch = 1, int width = 16)
{
	transfer_D2H();
	std::cout << "**" << name << "\t: (" << size() << ")\t";
	std::cout << "n: " << shape_.n_ << " c: " << shape_.c_ << " h: " << shape_.h_ << " w: " << shape_.w_ << std::endl;
	std::cout << std::hex << "\t(host:" << h_data_ << ", device: " << d_data_ << ")" << std::dec << std::endl;

	if(view_param)
	{
		std::cout << std::fixed;
		std::cout.precision(6);

		int max_print_line = 4;
		if(width == 28)
		{
			std::cout.precision(3);
			max_print_line = 28;
		}

		int offset = 0;

		for(int n = 0; n < num_batch; n++)
		{
			if(num_batch > 1)
				std::cout << "<--- batch[" << n << "] --->" << std::endl;

			int count = 0;
			int print_line_count = 0;
			while( count < size() && print_line_count < max_print_line )
			{
				std::cout << "\t";
				for(int s = 0; s < width && count < size(); s++)
				{
					std::cout << h_data_.get()[size() * n + count + offset] << "\t";
					count++;
				}
				std::cout << std::endl;
				print_line_count++;
			}
		}

		std::cout.unsetf(std::ios::fixed);
	}
}



